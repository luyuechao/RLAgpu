#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>

#include <math.h>
#include <iostream>
#include <iomanip>
#include <fstream>

#include "gpuErrorCheck.h"
#include "rsvd.h"
#include "nccl.h"

/*
 ToDo list
 
 */
void enableGpuAccessPeer(int nDev, bool testAccess = false){
    
    int access;
    
    for (int i = 0; i < nDev; i++){
        CHECK_CUDA( hipSetDevice( i ) );
        for (int j = 0; j < nDev; j++){
            if(i != j){
                CHECK_CUDA( hipDeviceCanAccessPeer(&access, i, j) );
                if (access){
                    //printf("GPU %d can access GPU %d\n", i, j);
                    CHECK_CUDA( hipDeviceEnablePeerAccess(j, 0) );
                    //CHECK_CUDA( hipSetDevice(j) );
                    //CHECK_CUDA( hipDeviceEnablePeerAccess(i, 0) );
                    //CHECK_CUDA( hipSetDevice(i) );
                }else{
                    //printf("GPU %d can NOT access GPU %d\n", i, j);
                }
            }
        }
    }
    
    if (!testAccess) return;
    
    // test peer copy
    double *d_send[nDev];
    double *d_recv[nDev];
    hipStream_t cuStream[nDev];
    int test_size = 1024;
    
    for (int i = 0; i < nDev; ++i){
        CHECK_CUDA( hipSetDevice(i) );
        CHECK_CUDA( hipMalloc(&d_send[i],   test_size * sizeof(double)) );
        CHECK_CUDA( hipMalloc(&d_recv[i],   test_size * sizeof(double)) );
        CHECK_CUDA( hipMemset(d_send[i], 1, test_size * sizeof(double)) );
        CHECK_CUDA( hipMemset(d_recv[i], 0, test_size * sizeof(double)) );
        CHECK_CUDA( hipStreamCreate(&cuStream[i]) );
        
    }
    
    // copy data from device 0 to all other devices
    
    for (int i = 1; i < nDev; ++i){ // be careful of i=1
        CHECK_CUDA( hipSetDevice(i) );
        for (int j = 0; j < nDev; j++){
            if(i != j){
                CHECK_CUDA( hipMemcpyPeerAsync( d_recv[j], j,
                                                d_send[i], i,
                                                test_size * sizeof(double),
                                                cuStream[j] ) );
            }
        }
    }
    
    
    // copy send 0 to recv 1
    // CHECK_CUDA( hipSetDevice(0) );
    //CHECK_CUDA( hipMemcpyPeerAsync( d_recv[1], 1, d_send[0], 0, 1, cuStream[0] ) );
    
    // copy send 1 to recv 0
    // CHECK_CUDA( hipSetDevice(1) );
    // CHECK_CUDA( hipMemcpyPeerAsync( d_recv[0], 0, d_send[1], 1, 1, cuStream[1] ) );
    
    // cleanup
    for (int i = 0; i < nDev; ++i){
        
        CHECK_CUDA( hipSetDevice(i) );
        CHECK_CUDA( hipFree(d_send[i]) );
        CHECK_CUDA( hipFree(d_recv[i]) );
        CHECK_CUDA( hipStreamDestroy(cuStream[i]) );
        
    }
    printf("The system passed peer copy test.\n");
}

void powerIterationMulti_colSampling( double **Y, double **TEMP, double **Ac,
                                     const uint64_t m, const uint64_t nb, const uint64_t l,
                                     const uint64_t q, const uint64_t i,
                                     hipblasHandle_t *cublasH){
    
    const uint64_t ldAc = roundup_to_32X( m );
    const uint64_t ldY = ldAc;
    
    const double double_one = 1.0, double_zero = 0.0;
    
    if(q == 0){// no iteration
        return;
    }
    
    const uint64_t ldTEMP = roundup_to_32X( nb );
    CHECK_CUDA( hipMalloc((void**)&TEMP[i], ldTEMP * l * sizeof(double)) );
    
    for(uint64_t j = 0; j < q; j++){
        // TEMP = Ac' * Y, P[nsxl] = Ac'[nsxm] * Y[mxl]
        CHECK_CUBLAS( hipblasDgemm( cublasH[i], HIPBLAS_OP_T, HIPBLAS_OP_N,
                                  nb, l, m,
                                  &double_one,
                                  Ac[i],   ldAc,
                                  Y[i],   ldY,
                                  &double_zero,
                                  TEMP[i], ldTEMP) );
        
        //Y = Ac * TEMP, Y[mxl] = Ac[mxns] * TEMP[nsxl]
        CHECK_CUBLAS( hipblasDgemm( cublasH[i], HIPBLAS_OP_N, HIPBLAS_OP_N,
                                  m, l, nb,
                                  &double_one,
                                  Ac[i],   ldAc,
                                  TEMP[i], ldTEMP,
                                  &double_zero,
                                  Y[i],   ldY) );
        
    }

    CHECK_CUDA( hipFree(TEMP[i]) );
    
}

void brsvdMulti_colSampling(double *U, double *Sv, double *VT, double *d_A,
                            const uint64_t m, const uint64_t n, const uint64_t l, const uint64_t q,
                            const uint64_t batch,
                            ncclComm_t *comms, hipsolverHandle_t &cusolverH){
    
    uint64_t nb = 0, lastbatch = 0;
    
    if(batch == 0){
        nb = n;
        lastbatch = n;
    }else{
        nb = n / batch;
        lastbatch = n - nb * batch;
    }
    
    int nDev = batch;
    
    double    *Ac[nDev];
    double *Omega[nDev];
    double     *Y[nDev];
    double  *TEMP[nDev];
    
    //setup parameters
    const uint64_t ldAc    = roundup_to_32X( m ); // pad columns into multiple of 32
    const uint64_t ldOmega = roundup_to_32X( nb);
    const uint64_t ldY = ldAc;
    
    // double_one & double_zero for matrix multiplication
    const double double_one = 1.0, double_zero = 0.0;
    
    hiprandGenerator_t randGen[nDev];
    hipblasHandle_t    cublasH[nDev];
    hipStream_t     cuStream[nDev];
    
    // initialize hiprand and cuBLAS
    for (int i = 0; i < nDev; ++i){
        CHECK_CUDA( hipSetDevice(i) );
        CHECK_CUDA( hipMalloc(&Ac[i], ldAc * nb * sizeof(double)) );
        CHECK_CUDA( hipMalloc(&Y[i], ldY * l  * sizeof(double)) );
        // create hiprand handle
        CHECK_CURAND( hiprandCreateGenerator(&randGen[i], HIPRAND_RNG_PSEUDO_DEFAULT) );
        // seeds for hiprand
        CHECK_CURAND( hiprandSetPseudoRandomGeneratorSeed(randGen[i], time(NULL)) );
        // create cublas handle
        CHECK_CUBLAS( hipblasCreate(&cublasH[i]) );
        // create and set stream
        CHECK_CUDA( hipStreamCreate(&cuStream[i]) );
        CHECK_CURAND( hiprandSetStream(randGen[i], cuStream[i]) );
        CHECK_CUBLAS( hipblasSetStream(cublasH[i], cuStream[i]) );
        
    }
    
    CHECK_CUDA( hipDeviceSynchronize() );
    
    // print_device_matrix(d_A, m, n, ldAc, "A");
    // scatter A, generate random number, multiplication
    for (int i = 0; i < nDev; ++i){
        CHECK_CUDA( hipSetDevice(i) );
        // scatter A
        CHECK_CUDA( hipMemcpyPeerAsync(Ac[i], i,
                                        d_A + ldAc * nb * i, 0,
                                        ldAc * nb * sizeof(double), cuStream[i]) );
        CHECK_CUDA( hipMalloc(&Omega[i], ldOmega * l * sizeof(double)) );

        // generate double normal distribution with mean = 0.0, stddev = 1.0
        CHECK_CURAND( hiprandGenerateNormalDouble(randGen[i], Omega[i], ldOmega * l, 0.0, 1.0) );
        
        /*********** Y[mxl] = Ac[mxnb] * Omega[nbxl] ************/
        CHECK_CUBLAS( hipblasDgemm( cublasH[i],  HIPBLAS_OP_N, HIPBLAS_OP_N,
                                  m, l, nb,
                                  &double_one,
                                  Ac[i],    ldAc,
                                  Omega[i], ldOmega,
                                  &double_zero,
                                  Y[i],    ldY) );
        //printf("Ac[%d]", i);
        //print_device_matrix(Ac[i], m, nb, ldAc, " ");
        //printf("Y[%d]", i);
        //print_device_matrix(Y[i], m, l, ldY, " ");
        CHECK_CUDA( hipFree(Omega[i]) );
        
        /********** Step 2: power iteration *********/
        powerIterationMulti_colSampling(Y, TEMP, Ac, m, nb, l, q, i, cublasH);
    
    }
    

    // sum Y to Q
    double *Q[nDev];

    CHECK_CUDA( hipSetDevice(0) );
    int QR_workSpace = orth_CAQR_size(m, l);
    
    // Q[0] is larger than others for CAQR decomposition
    CHECK_CUDA( hipMalloc(&Q[0],        QR_workSpace * sizeof(double)) );
    CHECK_CUDA( hipMemsetAsync(Q[0], 0, QR_workSpace * sizeof(double)) );
    
    for (int i = 1; i < nDev; ++i){
        CHECK_CUDA( hipSetDevice(i) );
        CHECK_CUDA( hipMalloc(&Q[i],        ldY * l * sizeof(double)) );
        CHECK_CUDA( hipMemsetAsync(Q[i], 0, ldY * l * sizeof(double)) );
    }
    
    CHECK_NCCL( ncclGroupStart() );
    const int nccl_root = 0; // set root to dev 0
    for (int i = 0; i < nDev; ++i){
        CHECK_NCCL( ncclReduce((const void*)Y[i],
                               (void*)Q[i], ldY * l, ncclDouble,
                               ncclSum, nccl_root, comms[i], cuStream[i]) );
    }
    
    CHECK_NCCL( ncclGroupEnd() );
    
    //synchronizing on CUDA streams to wait for completion of NCCL operation
    for (int i = 0; i < nDev; ++i) {
        CHECK_CUDA( hipSetDevice(i) );
        CHECK_CUDA( hipStreamSynchronize(cuStream[i]) );
        CHECK_CUDA( hipFree(Y[i]) );
    }
    
    // device 0 to process last batch
    if(lastbatch != 0){
        int i = 0;
        CHECK_CUDA( hipSetDevice(i) );
        CHECK_CUDA( hipMalloc(&Y[i], ldY * l  * sizeof(double)) );
        CHECK_CUDA( hipMalloc((void**)&Omega[i], lastbatch * l * sizeof(double)) );
        CHECK_CURAND( hiprandGenerateNormalDouble(randGen[i], Omega[i], lastbatch * l, 0.0, 1.0));
        CHECK_CURAND( hiprandDestroyGenerator( randGen[i] ) );
        
        /***********  Y[mxl] = As[mxlastbatch] * Omega[lastbatchxl] ************/
        CHECK_CUBLAS( hipblasDgemm( cublasH[i],  HIPBLAS_OP_N, HIPBLAS_OP_N,
                                  m, l, lastbatch,
                                  &double_one,
                                  d_A + ldAc * nb * batch, ldAc,
                                  Omega[i], lastbatch,
                                  &double_one, // 1.0 (add to Y[0])
                                  Y[i], ldY) );
        
        CHECK_CUDA( hipFree(Omega[i]) );
        
        if(q == 0){// no iteration
            return;
        }
        const uint64_t ldTEMP = roundup_to_32X( nb );
        CHECK_CUDA( hipMalloc((void**)&TEMP[i], ldTEMP * l * sizeof(double)) );
        
        for(uint64_t j = 0; j < q -1; j++){
            // TEMP = Ac' * Y, P[nsxl] = Ac'[nsxm] * Y[mxl]
            CHECK_CUBLAS( hipblasDgemm( cublasH[i], HIPBLAS_OP_T, HIPBLAS_OP_N,
                                      nb, l, m,
                                      &double_one,
                                      Ac[i],   ldAc,
                                      Y[i],   ldY,
                                      &double_zero,
                                      TEMP[i], ldTEMP) );
            
            //Y = Ac * TEMP, Y[mxl] = Ac[mxns] * TEMP[nsxl]
            CHECK_CUBLAS( hipblasDgemm( cublasH[i], HIPBLAS_OP_N, HIPBLAS_OP_N,
                                      m, l, nb,
                                      &double_one,
                                      Ac[i],   ldAc,
                                      TEMP[i], ldTEMP,
                                      &double_zero,
                                      Y[i],   ldY) );
            
        }
        // TEMP = Ac' * Y, P[nsxl] = Ac'[nsxm] * Y[mxl]
        CHECK_CUBLAS( hipblasDgemm( cublasH[i], HIPBLAS_OP_T, HIPBLAS_OP_N,
                                  nb, l, m,
                                  &double_one,
                                  Ac[i],   ldAc,
                                  Y[i],   ldY,
                                  &double_zero,
                                  TEMP[i], ldTEMP) );
        
        //Q = Q + Ac * TEMP, Y[mxl] = Ac[mxns] * TEMP[nsxl]
        CHECK_CUBLAS( hipblasDgemm( cublasH[i], HIPBLAS_OP_N, HIPBLAS_OP_N,
                                  m, l, nb,
                                  &double_one,
                                  Ac[i],   ldAc,
                                  TEMP[i], ldTEMP,
                                  &double_one, // one
                                  Q[i],   ldY) );
        
        CHECK_CUDA( hipFree(TEMP[i]) );
        
        CHECK_CUDA( hipFree(Y[i]) );
    }
    

    //QR on device 0
    CHECK_CUDA( hipSetDevice(0) );
    //print_device_matrix(Q[0], m, l, ldY, "Q");
    orth_CAQR(Q[0], m, l);
    
    //print_device_matrix(Q[0], m, l, ldY, "Q");
    const uint64_t ldB = roundup_to_32X( l );
    double *B[nDev];
    
    for (int i = 0; i < nDev; ++i){
        CHECK_CUDA( hipSetDevice(i) );
        // scatter Q
        if(i > 0){
            CHECK_CUDA( hipMemcpyPeerAsync(Q[i], i,
                                            Q[0], 0,
                                            ldY * l * sizeof(double), cuStream[i] ) );
            CHECK_CUDA( hipMalloc(&B[i], ldB * nb * sizeof(double)) );
        }else{ // i = 0
            CHECK_CUDA( hipMalloc(&B[i], ldB * n * sizeof(double)) );
        }
        
        // Bc = Q’ * Ac, Bc[lxnb] = Q'[lxm] * Ac[mxnb]
        CHECK_CUBLAS( hipblasDgemm( cublasH[i], HIPBLAS_OP_T, HIPBLAS_OP_N,
                                  l, nb, m,
                                  &double_one,
                                  Q[i],  ldY,
                                  Ac[i], ldAc,
                                  &double_zero,
                                  B[i], ldB) );
        
        // printf("B[%d]", i);
        // print_device_matrix(B[i], l, nb, ldB, " ");
        
        CHECK_CUDA( hipFree(Ac[i]) );
        
    }

    CHECK_CUDA( hipSetDevice(0) );

    
    // gather Bc to B on device 0
    for (int i = 1; i < nDev; ++i){
        CHECK_CUDA( hipMemcpyPeerAsync(B[0] + ldB * nb * i, 0,
                                        B[i], i,
                                        ldB * nb * sizeof(double), cuStream[i] ) );
    }
    
    if(lastbatch != 0){
        // Bc = Q’ * Ac, Bc[lxlastbatch] = Q'[lxm] * Ac[mxlastbatch]
        CHECK_CUBLAS( hipblasDgemm( cublasH[0], HIPBLAS_OP_T, HIPBLAS_OP_N,
                                  l, lastbatch, m,
                                  &double_one,
                                  Q[0], ldY,
                                  d_A + ldAc * nb * batch, ldAc,
                                  &double_zero,
                                  B[0] + ldB * nb * batch, ldB) );
    }
    
    //CHECK_CUDA( hipFree(d_A) );
    // print_device_matrix(B[0], l, n, ldB, "B");
    // transpose B
    const uint64_t ldBT = roundup_to_32X( n );
    double *BT;
    CHECK_CUDA( hipMalloc((void**)&BT, ldBT * l * sizeof(double)) );
    
    // synchronize the gather of B (necessary)
    for (int i = 0; i < nDev; ++i) {
        CHECK_CUDA( hipStreamSynchronize(cuStream[i]) );
    }
    
    transposeGPU(cublasH[0], BT, B[0], l, n);
    
    for (int i = 0; i < nDev; ++i){
        CHECK_CUDA( hipSetDevice(i) );
        CHECK_CUDA( hipFree(B[i]) );
    }
    
    CHECK_CUDA( hipSetDevice(0) );
    /********** Step 5: SVD on BT (nxl) *********/
    //  max memroy usage: B[lxn] + UhatT[lxl] + V[nxl] = l(2n+1)
    const uint64_t ldUhat = roundup_to_32X( l ), ldV = roundup_to_32X( n );
    
    /********** Step 5: SVD on BT (nxl) *********/
    double *UhatT, *V;
    CHECK_CUDA( hipMalloc((void**)&UhatT, ldUhat * l * sizeof(double)) );
    CHECK_CUDA( hipMalloc((void**)&V,     ldV    * l * sizeof(double)) );
    CHECK_CUDA( hipMemsetAsync(UhatT,0,   ldUhat * l * sizeof(double)) );
    CHECK_CUDA( hipMemsetAsync(V,    0,   ldV    * l * sizeof(double)) );
    
    //V[nxl] * Sv[lxl] * UhatT[lxl] = BT[nxl]
    svd(cusolverH, V, Sv, UhatT, BT, n, l);
    
    CHECK_CUDA( hipFree(BT) );
    
    /********** Step 6:  U = Q * Uhat, U[mxl] = Q[mxl] * Uhat[lxl] *********/
    CHECK_CUBLAS( hipblasDgemm( cublasH[0], HIPBLAS_OP_N, HIPBLAS_OP_T,
                              m, l, l,
                              &double_one,
                              Q[0], ldY,
                              UhatT, ldUhat,
                              &double_zero,
                              U, ldAc) );
    
    CHECK_CUDA( hipFree(UhatT) );

    /**********Step 7: transpose V ****/
    transposeGPU(cublasH[0], VT, V, n, l);
    CHECK_CUDA( hipFree(V) );
    
    // cleanup
    for (int i = 0; i < nDev; ++i){
        
        CHECK_CUDA( hipSetDevice(i) );
        CHECK_CUDA( hipFree(Q[i]) );
        CHECK_CUBLAS( hipblasDestroy(cublasH[i]) );
        CHECK_CUDA( hipStreamDestroy( cuStream[i] ) );
    }
    
    // go back device 0
    //CHECK_CUDA( hipSetDevice(0) );
    //CHECK_CUDA( hipMalloc((void**)&d_A,  ldAc * n * sizeof(double)) );
    //CHECK_CUDA( hipMemsetAsync(d_A, 0,   ldAc * n * sizeof(double)) );
    
}

void rsvd_multi_gpu(double *dev_U, double *dev_S, double *dev_VT, double *dev_A,
                    const uint64_t m, const uint64_t n, const uint64_t l, const uint64_t q,
                    hipsolverHandle_t &cusolverH){
    
    int nDev;
    CHECK_CUDA( hipGetDeviceCount(&nDev) );
    //printf("%d GPU in this system.\n", nDev);
    
    /*nDev = 1;
    enableGpuAccessPeer(nDev, true);
    int devs[nDev] = {0};
    */
    // initialize multi-GPU peer access
    enableGpuAccessPeer(nDev, false);
    int devs[nDev] = {0,1};
    
    // initializing NCCL
    ncclComm_t comms[nDev];
    CHECK_NCCL( ncclCommInitAll(comms, nDev, devs) );
    
    // main process
    brsvdMulti_colSampling(dev_U, dev_S, dev_VT, dev_A, m, n, l, q,
                           nDev, comms, cusolverH);
    
    CHECK_CUDA( hipSetDevice(0) );
}

